#include "hip/hip_runtime.h"
/* cuda pcg algorithm */

#include "cuda_pcg.h"
#include "csr_type.h"
#include "cuda_handler.h"
#include "cuda_malloc.h"
#include "cuda_lin_alg.h"
#include "cuda_wrapper.h"
#include "hip/hip_runtime_api.h"

#ifdef __cplusplus
extern "C" {
extern CUDA_Handle_t *CUDA_handle;
}
#endif

__global__ void scalar_division_kernel(c_float *res, const c_float *num, const c_float *den)
{
    *res = (*num) / (*den);
}

/* computes:  d_y = (P + sigma*I + A'*R*A) * d_x */
static void mat_vec_prod(cudapcg_solver *s, c_float *d_y, const c_float  *d_x, c_int device)
{

    c_float *sigma;
    c_float H_0 = 0.0;
    c_float H_1  = 1.0;
    c_int n = s->n;
    c_int m = s->m;
    csr *P  = s->P;
    csr *A  = s->A;
    csr *At = s->At;

    if (device)
    {
        sigma = s->d_sigma;
    }
    else
    {
        sigma = s->h_sigma;
    }

    /* d_y = d_x */
    checkCudaErrors(hipMemcpy(d_y, d_x, n * sizeof(c_float), hipMemcpyDeviceToDevice));

    /* d_y *= sigma */
    checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, n, sigma, d_y, 1));

    /* d_y += P * d_x */
    checkCudaErrors(cusparseCsrmv(CUDA_handle->cusparseHandle, P->alg, P->m, P->n, P->nnz, &H_1, P->MatDescription, P->val, P->row_ptr, P->col_ind, d_x, &H_1, d_y, P->buffer));

    if (m == 0) return;

    if (!s->d_rho_vec)
    {
        /* d_z = rho * A * d_x */
        checkCudaErrors(cusparseCsrmv(CUDA_handle->cusparseHandle, A->alg, A->m, A->n, A->nnz, s->h_rho, A->MatDescription, A->val, A->row_ptr, A->col_ind, d_x, &H_0, s->d_z, A->buffer));
    }
    else
    {
        /* d_z = A * d_x */
        checkCudaErrors(cusparseCsrmv(CUDA_handle->cusparseHandle, A->alg, A->m, A->n, A->nnz, &H_1, A->MatDescription, A->val, A->row_ptr, A->col_ind, d_x, &H_0, s->d_z, A->buffer));

        /* d_z = diag(d_rho_vec) * dz */
        cuda_vec_ew_prod(s->d_z, s->d_z, s->d_rho_vec, m);
    }

    /* d_y += A' * d_z */
    checkCudaErrors(cusparseCsrmv(CUDA_handle->cusparseHandle, At->alg, At->m, At->n, At->nnz, &H_1, At->MatDescription, At->val, At->row_ptr, At->col_ind, s->d_z, &H_1, d_y, A->buffer));
}

/* pcg algorithm */
c_int cuda_pcg(cudapcg_solver *s, c_float eps, c_int max_niter)
{

    c_float *ptr_tmp;
    c_int niter = 0;
    c_int n = s->n;
    c_float H_m_1 = -1.0;

    /* set up problem */

    if (!s->warm_start)
    {
        /* d_x = 0 */
        checkCudaErrors(hipMemset(s->d_x, 0, n * sizeof(c_float)));
    }

    /* d_p = 0 */
    checkCudaErrors(hipMemset(s->d_p, 0, n * sizeof(c_float)));

    /* d_r = K * d_x */
    mat_vec_prod(s, s->d_r, s->d_x, 0);

    /* d_r -= d_rhs */
    checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, &H_m_1, s->d_rhs, 1, s->d_r, 1));

    /* h_r_norm = |d_r| */
    s->vector_norm(s->d_r, n, s->h_r_norm);

    /* need to change CUBLAS mode */
    hipblasSetPointerMode(CUDA_handle->cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);

    if (s->precondition)
    {
        /* d_y = M \ d_r */
        cuda_vec_ew_prod(s->d_y, s->d_diag_precond_inv, s->d_r, n);
    }

    /* d_p = -d_y */
    checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, s->D_MINUS_ONE, s->d_y, 1, s->d_p, 1));

    /* rTy = d_r' * d_y */
    checkCudaErrors(cublasTdot(CUDA_handle->cublasHandle, n, s->d_y, 1, s->d_r, 1, s->rTy));

    /* synchronize for timing */
    hipDeviceSynchronize();

    /* Run the PCG algorithm */
    while ( *(s->h_r_norm) > eps && niter < max_niter )
    {

        /* d_Kp = K * d_p */
        mat_vec_prod(s, s->d_Kp, s->d_p, 1);

        /* pKp = d_p' * d_Kp */
        checkCudaErrors(cublasTdot(CUDA_handle->cublasHandle, n, s->d_p, 1, s->d_Kp, 1, s->pKp));

        /* alpha = rTy / pKp */
        scalar_division_kernel<<<1,1>>>(s->alpha, s->rTy, s->pKp);

        /* d_x += alpha * d_p */
        checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, s->alpha, s->d_p, 1, s->d_x, 1));

        /* d_r += alpha * d_Kp */
        checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, s->alpha, s->d_Kp, 1, s->d_r, 1));

        if (s->precondition)
        {
            /* d_y = M \ d_r */
            cuda_vec_ew_prod(s->d_y, s->d_diag_precond_inv, s->d_r, n);
        }

        /* Swap pointers to rTy and rTy_prev */
        ptr_tmp = s->rTy_prev;
        s->rTy_prev = s->rTy;
        s->rTy = ptr_tmp;

        /* rTy = d_r' * d_y */
        checkCudaErrors(cublasTdot(CUDA_handle->cublasHandle, n, s->d_y, 1, s->d_r, 1, s->rTy));

        /* Update residual norm */
        s->vector_norm(s->d_r, n, s->d_r_norm);
        checkCudaErrors(hipMemcpyAsync(s->h_r_norm, s->d_r_norm, sizeof(c_float), hipMemcpyDeviceToHost));

        /* beta = rTy / rTy_prev */
        scalar_division_kernel<<<1,1>>>(s->beta, s->rTy, s->rTy_prev);

        /* d_p *= beta */
        checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, n, s->beta, s->d_p, 1));

        /* d_p -= d_y */
        checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, s->D_MINUS_ONE, s->d_y, 1, s->d_p, 1));

        hipDeviceSynchronize();
        niter++;

    } /* End of the PCG algorithm */

    /* change CUBLAS pointer mode back */
    hipblasSetPointerMode(CUDA_handle->cublasHandle, HIPBLAS_POINTER_MODE_HOST);

    return niter;
}

/* update preconditioning  */
void cuda_pcg_update_precond(cudapcg_solver *s, c_int P_updated, c_int A_updated, c_int R_updated)
{

    void    *buffer;
    c_float *mem_tmp;
    c_int    n  = s->n;
    csr     *At = s->At;

    size_t buff_size = n * (sizeof(c_float) + sizeof(c_int));

    if (!P_updated && !A_updated && !R_updated) return;

    if (P_updated)
    {
        /* Update d_P_diag_val */
        checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, n, s->P->val, s->d_P_diag_val, s->d_P_diag_ind, HIPSPARSE_INDEX_BASE_ZERO));
    }

    if (A_updated || R_updated)
    {
        /* Allocate memory */
        cuda_malloc((void **) &mem_tmp, At->nnz * sizeof(c_float));
        cuda_malloc((void **) &buffer, buff_size);

        /* Update d_AtRA_diag_val */
        if (!s->d_rho_vec)
        {   /* R = rho*I  -->  A'*R*A = rho * A'*A */

            if (A_updated)
            {
                /* Update d_AtA_diag_val */
                cuda_vec_ew_prod(mem_tmp, At->val, At->val, At->nnz);
                cuda_vec_segmented_sum(mem_tmp, At->row_ind, s->d_AtA_diag_val, buffer, n, At->nnz);
            }

            /* d_AtRA_diag_val = rho * d_AtA_diag_val */
            cuda_vec_add_scaled(s->d_AtRA_diag_val, s->d_AtA_diag_val, NULL, *s->h_rho, 0.0, n);
        }
        else
        {   /* R = diag(d_rho_vec)  -->  A'*R*A = A' * diag(d_rho_vec) * A */
            cuda_mat_rmult_diag_new(At, mem_tmp, s->d_rho_vec);   /* mem_tmp = A' * R */
            cuda_vec_ew_prod(mem_tmp, mem_tmp, At->val, At->nnz);     /* mem_tmp = mem_tmp * A */
            cuda_vec_segmented_sum(mem_tmp, At->row_ind, s->d_AtRA_diag_val, buffer, n, At->nnz);
        }

        cuda_free((void **) &mem_tmp);
        cuda_free((void **) &buffer);
    }

    /* d_diag_precond = sigma */
    cuda_vec_set_sc(s->d_diag_precond, *s->h_sigma, n);

    /* d_diag_precond += d_P_diag_val + d_AtRA_diag_val */
    cuda_vec_add_scaled3(s->d_diag_precond, s->d_diag_precond, s->d_P_diag_val, s->d_AtRA_diag_val, 1.0, 1.0, 1.0, n);

    /* d_diag_precond_inv = 1 / d_diag_precond */
    cuda_vec_reciprocal(s->d_diag_precond_inv, s->d_diag_precond, n);
}

#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019 ETH Zurich, Automatic Control Lab, Michel Schubiger, Goran Banjac.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

 #include "./include/cuda_lin_alg.h"
//  #include "cuda_configure.h"
//  #include "cuda_handler.h"
//  #include "cuda_malloc.h"
//  #include "cuda_wrapper.h"
//  #include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */
 
//  #include "csr_type.h"
// //  #include "glob_opts.h"

 
//  #include <thrust/reduce.h>
//  #include <thrust/execution_policy.h>

 
 
 /*******************************************************************************
  *                              GPU Kernels                                    *
  *******************************************************************************/
 
  __global__ void vec_set_sc_kernel(float *a,
                                    float  sc,
                                    int    n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     a[i] = sc;
   }
 }
 
 __global__ void vec_set_sc_cond_kernel(float     *a,
                                        const int *test,
                                        float      sc_if_neg,
                                        float      sc_if_zero,
                                        float      sc_if_pos,
                                        int        n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     if (test[i] == 0)      a[i] = sc_if_zero;
     else if (test[i] > 0)  a[i] = sc_if_pos;
     else                   a[i] = sc_if_neg;
   }
 }
 
 __global__ void vec_prod_pos_kernel(const float *a,
                                     const float *b,
                                     float       *res,
                                     int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   float res_kernel = 0.0;
 
   for(int i = idx; i < n; i += grid_size) {
     res_kernel += a[i] * c_max(b[i], 0.0);
   }
   atomicAdd(res, res_kernel);
 }
 
 __global__ void vec_prod_neg_kernel(const float *a,
                                     const float *b,
                                     float       *res,
                                     int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   float res_kernel = 0.0;
 
   for(int i = idx; i < n; i += grid_size) {
     res_kernel += a[i] * c_min(b[i], 0.0);
   }
   atomicAdd(res, res_kernel);
 }
 
 __global__ void vec_ew_prod_kernel(float       *c,
                                    const float *a,
                                    const float *b,
                                    int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
//  #ifdef DFLOAT
    //  c[i] = __fmul_rn(a[i], b[i]);
//  #else
     c[i] = __dmul_rn(a[i], b[i]);
//  #endif
   }
 }
 
 __global__ void vec_leq_kernel(const float *l,
                                const float *u,
                                int         *res,
                                int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     if (l[i] > u[i]) atomicAnd(res, 0);
   }
 }
 
 __global__ void vec_bound_kernel(float       *x,
                                  const float *z,
                                  const float *l,
                                  const float *u,
                                  int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     x[i] = c_min(c_max(z[i], l[i]), u[i]);
   }
 }
 
 __global__ void vec_project_polar_reccone_kernel(float       *y,
                                                  const float *l,
                                                  const float *u,
                                                  float        infval,
                                                  int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     if (u[i] > +infval) {
       if (l[i] < -infval) {
         /* Both bounds infinite */
         y[i] = 0.0;
       }
       else {
         /* Only upper bound infinite */
         y[i] = c_min(y[i], 0.0);
       }
     }
     else if (l[i] < -infval) {
       /* Only lower bound infinite */
       y[i] = c_max(y[i], 0.0);
     }
   }
 }
 
 __global__ void vec_in_reccone_kernel(const float *y,
                                       const float *l,
                                       const float *u,
                                       float        infval,
                                       float        tol,
                                       int         *res,
                                       int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     if ( (u[i] < +infval && y[i] > +tol) ||
          (l[i] > -infval && y[i] < -tol) )
       atomicAnd(res, 0);
   }
 }
 
 __global__ void vec_reciprocal_kernel(float       *b,
                                       const float *a,
                                       int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
//  #ifdef DFLOAT
    //  b[i] = __frcp_rn(a[i]);
//  #else
     b[i] = __drcp_rn(a[i]);
//  #endif
   }
 }
 
 __global__ void vec_sqrt_kernel(float *a,
                                 int    n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
//  #ifdef DFLOAT
     a[i] = __fsqrt_rn(a[i]);
//  #else
     a[i] = __dsqrt_rn(a[i]);
//  #endif
   }
 }
 
 __global__ void vec_max_kernel(float       *c,
                                const float *a,
                                const float *b,
                                int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     c[i] = c_max(a[i], b[i]);
   }
 }
 
 __global__ void vec_min_kernel(float       *c,
                                const float *a,
                                const float *b,
                                int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     c[i] = c_min(a[i], b[i]);
   }
 }
 
 __global__ void vec_bounds_type_kernel(int         *iseq,
                                        const float *l,
                                        const float *u,
                                        float        infval,
                                        float        tol,
                                        int         *has_changed,
                                        int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     if (u[i] - l[i] < tol) {
       /* Equality constraints */
       if (iseq[i] != 1) {
         iseq[i] = 1;
         atomicOr(has_changed, 1);
       }
     }
     else if ( (l[i] < -infval) && (u[i] > infval) ) {
       /* Loose bounds */
       if (iseq[i] != -1) {
         iseq[i] = -1;
         atomicOr(has_changed, 1);
       }
     }
     else {
       /* Inequality constraints */
       if (iseq[i] != 0) {
         iseq[i] = 0;
         atomicOr(has_changed, 1);
       }
     }
   }
 }
 
 __global__ void vec_set_sc_if_lt_kernel(float       *x,
                                         const float *z,
                                         float        testval,
                                         float        newval,
                                         int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     x[i] = z[i] < testval ? newval : z[i];
   }
 }
 
 __global__ void vec_set_sc_if_gt_kernel(float       *x,
                                         const float *z,
                                         float        testval,
                                         float        newval,
                                         int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     x[i] = z[i] > testval ? newval : z[i];
   }
 }
 
 __global__ void mat_lmult_diag_kernel(const int   *row_ind,
                                       const float *diag,
                                       float       *data,
                                       int          nnz) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < nnz; i += grid_size) {
     int row = row_ind[i];
     data[i] *= diag[row];
   }
 }
 
 __global__ void mat_rmult_diag_kernel(const int   *col_ind,
                                       const float *diag,
                                       float       *data,
                                       int          nnz) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < nnz; i += grid_size) {
     int column = col_ind[i];
     data[i] *= diag[column];
   }
 }
 
 __global__ void mat_rmult_diag_new_kernel(const int   *col_ind,
                                           const float *diag,
                                           const float *data_in,
                                           float       *data_out,
                                           int          nnz) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < nnz; i += grid_size) {
     int column = col_ind[i];
     data_out[i] = data_in[i] * diag[column];
   }
 }
 
 __global__ void vec_abs_kernel(float *a,
                                int    n) {
 
   int i  = threadIdx.x + blockDim.x * blockIdx.x;
 
   if (i < n) {
//  #ifdef DFLOAT
     a[i] = fabsf(a[i]);
//  #else
     a[i] = fabs(a[i]);
//  #endif
   }
 }
 
 __global__ void scatter_kernel(float       *out,
                                const float *in,
                                const int   *ind,
                                int          n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     int j = ind[i];
     out[j] = in[i];
   }
 }
 
 /*
  * This code complements the cublasITamax routine which only returns the 
  * one-based index to the maximum absolute value in d_x. 
 */
 __global__ void abs_kernel(const int   *index_one_based,
                            const float *d_x,
                            float       *res) {
 
   /* cublasITamax returns one-based index */
   (*res) = abs(d_x[(*index_one_based)-1]);
 }
 
 
 /*******************************************************************************
  *                         Private functions                                   *
  *******************************************************************************/
 
 /*
  *  out[j] = in[i], where j = ind[i] for i in [0,n-1]
  */
 void scatter(float       *out,
              const float *in,
              const int   *ind,
              int          n) {
 
   int num_blocks = (n / THREADS_PER_BLOCK) + 1;
   scatter_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(out, in, ind, n);
 }
 
 
 /*******************************************************************************
  *                          Thrust-related functions                           *
  *******************************************************************************/
 
 template<typename BinaryFunction>
 void Segmented_reduce(const int    *key_start,
                       int           number_of_keys,
                       int           num_segments,
                       const float  *values,
                       void           *buffer,
                       float        *result,
                       BinaryFunction  binary_op) {
  
   int num_nnz_rows;
 
  /*  Memory layout of buffer:
   *  [ m*sizeof(float) Bytes | m*sizeof(int) Bytes]
   *  where m = "number of rows"
   */
   float *intermediate_result = (float*) buffer; 
   int   *nnz_rows            = (int*) (&intermediate_result[num_segments]);
 
   thrust::pair<int*,float*> new_end;
   thrust::equal_to<int> binary_pred;
   
   new_end = thrust::reduce_by_key(thrust::device,
                                   key_start,
                                   key_start + number_of_keys,
                                   values,
                                   nnz_rows,
                                   intermediate_result,
                                   binary_pred,
                                   binary_op);
 
   num_nnz_rows = new_end.first - nnz_rows;
   checkCudaErrors(hipMemset(result, 0, num_segments * sizeof(float)));
   scatter(result, intermediate_result, nnz_rows, num_nnz_rows);
 }
 
 template<typename T>
 struct abs_maximum {
   typedef T first_argument_type;
   typedef T second_argument_type;
   typedef T result_type;
   __host__ __device__ T operator()(const T &lhs, const T &rhs) const {return max(abs(lhs), abs(rhs));}
  };
 
 template void Segmented_reduce<abs_maximum<float>>(const int          *key_start,
                                                      int                 number_of_keys,
                                                      int                 number_of_segments,
                                                      const float        *values,
                                                      void                 *buffer,
                                                      float              *result,
                                                      abs_maximum<float>  binary_op);
 
 
 /*******************************************************************************
  *                           API Functions                                     *
  *******************************************************************************/
 
 void cuda_vec_copy_d2d(float       *d_y,
                        const float *d_x,
                        int          n) {
 
   checkCudaErrors(hipMemcpy(d_y, d_x, n * sizeof(float), hipMemcpyDeviceToDevice));
 }
 
 void cuda_vec_copy_h2d(float       *d_y,
                        const float *h_x,
                        int          n) {
 
   checkCudaErrors(hipMemcpy(d_y, h_x, n * sizeof(float), hipMemcpyHostToDevice));
 }
 
 void cuda_vec_copy_d2h(float       *h_y,
                        const float *d_x,
                        int          n) {
 
   checkCudaErrors(hipMemcpy(h_y, d_x, n * sizeof(float), hipMemcpyDeviceToHost));
 }
 
 void cuda_veint_copy_h2d(int       *d_y,
                            const int *h_x,
                            int        n) {
 
   checkCudaErrors(hipMemcpy(d_y, h_x, n * sizeof(int), hipMemcpyHostToDevice));
 }
 
 void cuda_veint_copy_d2h(int       *h_y,
                            const int *d_x,
                            int        n) {
 
   checkCudaErrors(hipMemcpy(h_y, d_x, n * sizeof(int), hipMemcpyDeviceToHost));
 }
 
 void cuda_vec_set_sc(float *d_a,
                      float  sc,
                      int    n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
   vec_set_sc_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_a, sc, n);
 }
 
 void cuda_vec_set_sc_cond(float     *d_a,
                           const int *d_test,
                           float      sc_if_neg,
                           float      sc_if_zero,
                           float      sc_if_pos,
                           float      n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_set_sc_cond_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_a, d_test, sc_if_neg, sc_if_zero, sc_if_pos, n);
 }
 
 void cuda_vec_mult_sc(float *d_a,
                       float  sc,
                       int    n) {
 
   checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, n, &sc, d_a, 1));
 }
 
 void cuda_vec_add_scaled(float       *d_x,
                          const float *d_a,
                          const float *d_b,
                          float        sca,
                          float        scb,
                          int          n) {
 
   if (d_x != d_a || sca != 1.0) {
     if (sca == 1.0) {
       /* d_x = d_a */
       checkCudaErrors(hipMemcpy(d_x, d_a, n * sizeof(float), hipMemcpyDeviceToDevice));
     }
     else if (d_x == d_a) {
       /* d_x *= sca */
       checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, n, &sca, d_x, 1));
     }
     else {
       /* d_x = 0 */
       checkCudaErrors(hipMemset(d_x, 0, n * sizeof(float)));
 
       /* d_x += sca * d_a */
       checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, &sca, d_a, 1, d_x, 1));
     }
   }
 
   /* d_x += scb * d_b */
   checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, &scb, d_b, 1, d_x, 1));
 }
 
 void cuda_vec_add_scaled3(float       *d_x,
                           const float *d_a,
                           const float *d_b,
                           const float *d_c,
                           float        sca,
                           float        scb,
                           float        scc,
                           int          n) {
 
   if (d_x != d_a || sca != 1.0) {
     if (sca == 1.0) {
       /* d_x = d_a */
       checkCudaErrors(hipMemcpy(d_x, d_a, n * sizeof(float), hipMemcpyDeviceToDevice));
     }
     else if (d_x == d_a) {
       /* d_x *= sca */
       checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, n, &sca, d_x, 1));
     }
     else {
       /* d_x = 0 */
       checkCudaErrors(hipMemset(d_x, 0, n * sizeof(float)));
 
       /* d_x += sca * d_a */
       checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, &sca, d_a, 1, d_x, 1));
     }
   }
 
   /* d_x += scb * d_b */
   checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, &scb, d_b, 1, d_x, 1));
 
   /* d_x += scc * d_c */
   checkCudaErrors(cublasTaxpy(CUDA_handle->cublasHandle, n, &scc, d_c, 1, d_x, 1));
 }
 
 void cuda_vec_norm_inf(const float *d_x,
                        int          n,
                        float       *h_res) {
 
   hipblasPointerMode_t mode;
   checkCudaErrors(hipblasGetPointerMode(CUDA_handle->cublasHandle, &mode));
 
   if (mode == HIPBLAS_POINTER_MODE_DEVICE) {
     checkCudaErrors(cublasITamax(CUDA_handle->cublasHandle, n, d_x, 1, CUDA_handle->d_index));
     abs_kernel<<<1,1>>>(CUDA_handle->d_index, d_x, h_res);  /* d_res actually */
   }
   else {
     int idx;
     checkCudaErrors(cublasITamax(CUDA_handle->cublasHandle, n, d_x, 1, &idx));
     checkCudaErrors(hipMemcpy(h_res, d_x + (idx-1), sizeof(float), hipMemcpyDeviceToHost));
     (*h_res) = abs(*h_res);
   }
 }
 
 void cuda_vec_norm_1(const float *d_x,
                      int          n,
                      float       *h_res) {
 
   cublasTasum(CUDA_handle->cublasHandle, n, d_x, 1, h_res);
 }
 
 void cuda_vec_norm_2(const float *d_x,
                      int          n,
                      float       *h_res) {
 
   cublasTnrm2(CUDA_handle->cublasHandle, n, d_x, 1, h_res);
 }
 
 void cuda_vec_scaled_norm_inf(const float *d_S,
                               const float *d_v,
                               int          n,
                               float       *h_res) {
 
   float *d_v_scaled;
 
   cuda_malloc((void **) &d_v_scaled, n * sizeof(float));
 
   /* d_v_scaled = d_S * d_v */
   cuda_vec_ew_prod(d_v_scaled, d_S, d_v, n);
 
   /* (*h_res) = |d_v_scaled|_inf */
   cuda_vec_norm_inf(d_v_scaled, n, h_res);
 
   cuda_free((void **) &d_v_scaled);
 }
 
 void cuda_vec_diff_norm_inf(const float *d_a,
                             const float *d_b,
                             int          n,
                             float       *h_res) {
 
   float *d_diff;
 
   cuda_malloc((void **) &d_diff, n * sizeof(float));
 
   /* d_diff = d_a - d_b */
   cuda_vec_add_scaled(d_diff, d_a, d_b, 1.0, -1.0, n);
 
   /* (*h_res) = |d_diff|_inf */
   cuda_vec_norm_inf(d_diff, n, h_res);
 
   cuda_free((void **) &d_diff);
 }
 
 void cuda_vec_mean(const float *d_x,
                    int          n,
                    float       *h_res) {
 
   cublasTasum(CUDA_handle->cublasHandle, n, d_x, 1, h_res);
   (*h_res) /= n;
 }
 
 void cuda_vec_prod(const float *d_a,
                    const float *d_b,
                    int          n,
                    float       *h_res) {
 
   checkCudaErrors(cublasTdot(CUDA_handle->cublasHandle, n, d_a, 1, d_b, 1, h_res));
 }
 
 void cuda_vec_prod_signed(const float *d_a,
                           const float *d_b,
                           int          sign,
                           int          n,
                           float       *h_res) {
 
   float *d_res;
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   cuda_calloc((void **) &d_res, sizeof(float));
 
   if (sign == 1) {
     vec_prod_pos_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_res, n);
     checkCudaErrors(hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost));
   }
   else if (sign == -1) {
     vec_prod_neg_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_res, n);
     checkCudaErrors(hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost));
   }
   else {
     checkCudaErrors(cublasTdot(CUDA_handle->cublasHandle, n, d_a, 1, d_b, 1, h_res));
   }
 
   cuda_free((void **) &d_res);
 }
 
 void cuda_vec_ew_prod(float       *d_c,
                       const float *d_a,
                       const float *d_b,
                       int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_ew_prod_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_c, d_a, d_b, n);
 }
 
 void cuda_vec_leq(const float *d_l,
                    const float *d_u,
                    int          n,
                    int         *h_res) {
 
   int *d_res;
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   cuda_malloc((void **) &d_res, sizeof(int));
 
   /* Initialize d_res to 1 */
   *h_res = 1;
   checkCudaErrors(hipMemcpy(d_res, h_res, sizeof(int), hipMemcpyHostToDevice));
 
   vec_leq_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_l, d_u, d_res, n);
 
   checkCudaErrors(hipMemcpy(h_res, d_res, sizeof(int), hipMemcpyDeviceToHost));
 
   cuda_free((void **) &d_res);
 }
 
 void cuda_vec_bound(float       *d_x,
                     const float *d_z,
                     const float *d_l,
                     const float *d_u,
                     int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_bound_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_x, d_z, d_l, d_u, n);
 }
 
 void cuda_vec_project_polar_reccone(float       *d_y,
                                     const float *d_l,
                                     const float *d_u,
                                     float        infval,
                                     int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_project_polar_reccone_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_y, d_l, d_u, infval, n);
 }
 
 void cuda_vec_in_reccone(const float *d_y,
                          const float *d_l,
                          const float *d_u,
                          float        infval,
                          float        tol,
                          int          n,
                          int         *h_res) {
 
   int *d_res;
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   cuda_malloc((void **) &d_res, sizeof(int));
 
   /* Initialize d_res to 1 */
   *h_res = 1;
   checkCudaErrors(hipMemcpy(d_res, h_res, sizeof(int), hipMemcpyHostToDevice));
 
   vec_in_reccone_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_y, d_l, d_u, infval, tol, d_res, n);
 
   checkCudaErrors(hipMemcpy(h_res, d_res, sizeof(int), hipMemcpyDeviceToHost));
 
   cuda_free((void **) &d_res);
 }
 
 void cuda_vec_reciprocal(float       *d_b,
                          const float *d_a,
                          int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_reciprocal_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_b, d_a, n);
 }
 
 void cuda_vec_sqrt(float *d_a,
                    int    n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_sqrt_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_a, n);
 }
 
 void cuda_vec_max(float       *d_c,
                   const float *d_a,
                   const float *d_b,
                   int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_max_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_c, d_a, d_b, n);
 }
 
 void cuda_vec_min(float       *d_c,
                   const float *d_a,
                   const float *d_b,
                   int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_min_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_c, d_a, d_b, n);
 }
 
 void cuda_vec_bounds_type(int         *d_iseq,
                           const float *d_l,
                           const float *d_u,
                           float        infval,
                           float        tol,
                           int          n,
                           int         *h_has_changed) {
 
   int *d_has_changed;
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   /* Initialize d_has_changed to zero */
   cuda_calloc((void **) &d_has_changed, sizeof(int));
 
   vec_bounds_type_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_iseq, d_l, d_u, infval, tol, d_has_changed, n);
 
   checkCudaErrors(hipMemcpy(h_has_changed, d_has_changed, sizeof(int), hipMemcpyDeviceToHost));
 
   cuda_free((void **) &d_has_changed);
 }
 
 void cuda_vec_set_sc_if_lt(float       *d_x,
                            const float *d_z,
                            float        testval,
                            float        newval,
                            int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_set_sc_if_lt_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_x, d_z, testval, newval, n);
 }
 
 void cuda_vec_set_sc_if_gt(float       *d_x,
                            const float *d_z,
                            float        testval,
                            float        newval,
                            int          n) {
 
   int number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
 
   vec_set_sc_if_gt_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(d_x, d_z, testval, newval, n);
 }
 
 void cuda_vec_segmented_sum(const float *d_values,
                             const int   *d_keys,
                             float       *d_res,
                             void          *d_buffer,
                             int          num_segments,
                             int          num_elements) {
 
   thrust::plus<float> binary_op;
   Segmented_reduce(d_keys, num_elements, num_segments, d_values, d_buffer, d_res, binary_op);
 }
 
 void cuda_mat_mult_sc(csr     *S,
                       csr     *At,
                       int    symmetric,
                       float  sc) {
 
   checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, S->nnz, &sc, S->val, 1));
 
   if (!symmetric) {
     /* Update At as well */
     checkCudaErrors(cublasTscal(CUDA_handle->cublasHandle, At->nnz, &sc, At->val, 1));
   }
 }
 
 void cuda_mat_lmult_diag(csr           *S,
                          csr           *At,
                          int          symmetric,
                          const float *d_diag) {
 
   int nnz = S->nnz;
   int number_of_blocks = (nnz / THREADS_PER_BLOCK) / ELEMENTS_PER_THREAD + 1;
 
   mat_lmult_diag_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(S->row_ind, d_diag, S->val, nnz);
 
   if (!symmetric) {
     /* Multiply At from right */
     mat_rmult_diag_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(At->col_ind, d_diag, At->val, nnz);
   }
 }
 
 void cuda_mat_rmult_diag(csr           *S,
                          csr           *At,
                          int          symmetric,
                          const float *d_diag) {
 
   int nnz = S->nnz;
   int number_of_blocks = (nnz / THREADS_PER_BLOCK) / ELEMENTS_PER_THREAD + 1;
 
   mat_rmult_diag_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(S->col_ind, d_diag, S->val, nnz);
 
   if (!symmetric) {
     /* Multiply At from left */
     mat_lmult_diag_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(At->row_ind, d_diag, At->val, nnz);
   }
 }
 
 void cuda_mat_rmult_diag_new(const csr     *S,
                              float       *d_buffer,
                              const float *d_diag) {
 
   int nnz = S->nnz;
   int number_of_blocks = (nnz / THREADS_PER_BLOCK) / ELEMENTS_PER_THREAD + 1;
 
   mat_rmult_diag_new_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(S->col_ind, d_diag, S->val, d_buffer, nnz);
 }
 
 void cuda_mat_Axpy(const csr     *A,
                    const float *d_x,
                    float       *d_y,
                    float        alpha,
                    float        beta) {
 
   if (A->nnz == 0 || alpha == 0.0) {
     /* d_y = beta * d_y */
     cuda_vec_mult_sc(d_y, beta, A->m);
     return;
   }
 
   checkCudaErrors(cusparseCsrmv(CUDA_handle->cusparseHandle, A->alg, A->m, A->n, A->nnz, &alpha, A->MatDescription, A->val, A->row_ptr, A->col_ind, d_x, &beta, d_y, A->buffer));
 }
 
 void cuda_mat_quad_form(const csr     *P,
                         const float *d_x,
                         float       *h_res) {
 
   int n = P->n;
   float *d_Px;
 
   cuda_malloc((void **) &d_Px, n * sizeof(float));
 
   /* d_Px = P * x */
   cuda_mat_Axpy(P, d_x, d_Px, 1.0, 0.0);
 
   /* h_res = d_Px' * d_x */
   cuda_vec_prod(d_Px, d_x, n, h_res);
 
   /* h_res *= 0.5 */
   (*h_res) *= 0.5;
 
   cuda_free((void **) &d_Px);
 }
 
 void cuda_mat_row_norm_inf(const csr *S,
                            float   *d_res) {
 
   int nnz      = S->nnz;
   int num_rows = S->m;
 
   if (nnz == 0) return;
 
   abs_maximum<float> binary_op;
   void *d_buffer;
   cuda_malloc(&d_buffer, num_rows * (sizeof(float) + sizeof(int)));
 
   /* 
   *  For rows with only one element, the element itself is returned.
   *  Therefore, we have to take the absolute value to get the inf-norm.
   */
   Segmented_reduce(S->row_ind, nnz, num_rows, S->val, d_buffer, d_res, binary_op);
   vec_abs_kernel<<<num_rows/THREADS_PER_BLOCK+1,THREADS_PER_BLOCK>>>(d_res, num_rows);
 
   cuda_free(&d_buffer);
 }
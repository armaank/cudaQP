#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019 ETH Zurich, Automatic Control Lab, Michel Schubiger, Goran Banjac.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

 #include "cuda_csr.h"
 #include "cuda_configure.h"
 #include "cuda_handler.h"
 #include "cuda_malloc.h"
 #include "cuda_wrapper.h"
 #include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */
 
 #include "csr_type.h"
 #include "glob_opts.h"
 
 #include <thrust/scan.h>
 #include <thrust/execution_policy.h>
  
 /* This function is implemented in cuda_lin_alg.cu */
 extern void scatter(float *out, const float *in, const int *ind, int n);
 
 
 /*******************************************************************************
  *                            GPU Kernels                                      *
  *******************************************************************************/
 
  /*
  * Expand an upper triangular matrix given in COO format to a symmetric
  * matrix. Each entry is duplicated with its column- and row index switched.
  * In the case of a diagonal element we set the indices to a value  that is
  * larger than n to easily remove it later. This is done to keep the memory
  * patern one to one (MAP operation).
  * 
  * Additionally, it adds additional n diagonal elements to have a full 
  * diagonal.
  * 
  * The output arrays row_ind_out and col_ind_out have to be of size 2*nnz+n.
  */
 __global__ void fill_full_matrix_kernel(int       *row_ind_out,
                                         int       *col_ind_out,
                                         int       *nnz_on_diag,
                                         int       *has_non_zero_diag_element,
                                         const int *__restrict__ row_ind_in,
                                         const int *__restrict__ col_ind_in,
                                         int        nnz,
                                         int        n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < nnz; i += grid_size) {
     int row = row_ind_in[i];
     int column = col_ind_in[i];
 
     row_ind_out[i] = row;
     col_ind_out[i] = column;
 
     if (row == column) {
       has_non_zero_diag_element[row] = 1;
       row_ind_out[i + nnz] = column + n; /* dummy value for sorting and removal later on */
       col_ind_out[i + nnz] = row + n;
       atomicAdd(nnz_on_diag, 1);
     }
     else {
       row_ind_out[i + nnz] = column;
       col_ind_out[i + nnz] = row;
     }
   }
 }
 
 /**
  * Insert elements at structural zeros on the diagonal of the sparse matrix
  * specified by row and column index (COO format). To keep a one-to-one memory
  * patern we add n new elements to the matrix. In case where there already is a
  * diagonal element we add a dummy entry. The dummy entries will be removed later.
  */
 __global__ void add_diagonal_kernel(int       *row_ind,
                                     int       *col_ind,
                                     const int *has_non_zero_diag_element,
                                     int        n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int row = idx; row < n; row += grid_size) {
     if (has_non_zero_diag_element[row] == 0) {
       row_ind[row] = row; 
       col_ind[row] = row;
     }
     else {
       row_ind[row] = row + n; /* dummy value, for easy removal after sorting */
       col_ind[row] = row + n;
     }
   }
 }
 
 /*
  * Permutation in: (size n, range 2*nnz+n):
  * 
  * Gathers from the following array to create the full matrix :
  * 
  *       |P_lower->val|P_lower->val|zeros(n)|
  *
  *       
  * Permutation out: (size n, range new_range)
  * 
  * Gathers from the following array to create the full matrix :
  * 
  *          |P_lower->val|zeros(1)|
  *                             
  *          | x[i] mod new_range    if x[i] <  2 * new_range
  * x[i] ->  | new_range             if x[i] >= 2 * new_range   
  * 
  */
 __global__ void reduce_permutation_kernel(int *permutation,
                                           int  new_range,
                                           int  n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for(int i = idx; i < n; i += grid_size) {
     if (permutation[i] < 2 * new_range) {
       permutation[i] = permutation[i] % new_range;
     }
     else {
       permutation[i] = new_range; /* gets the 0 element at nnz+1 of the value array */
     }
   }
 }
 
 __global__ void get_diagonal_indices_kernel(int *row_ind,
                                             int *col_ind,
                                             int  nnz,
                                             int *diag_index) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_size = blockDim.x * gridDim.x;
 
   for (int index = idx; index < nnz; index += grid_size) {
     int row = row_ind[index];
     int column = col_ind[index];
 
     if (row == column) {
       diag_index[row] = index;
     }
   }
 }
 
 __global__ void predicate_generator_kernel(const int *row_ind,
                                            const int *row_predicate,
                                            int       *predicate,
                                            int        nnz) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
   int grid_stride = gridDim.x * blockDim.x;
 
   for(int i = idx; i < nnz; i += grid_stride) {
     int row = row_ind[i];
     predicate[i] = row_predicate[row];
   }
 }
 
 template<typename T>
 __global__ void compact(const T *data_in,
                         T       *data_out,
                         int   *predicate,
                         int   *scatter_addres,
                         int    n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
 
   if(idx < n) {
     if(predicate[idx]) {
       int write_ind = scatter_addres[idx] - 1;
       data_out[write_ind] = data_in[idx];
     }
   }
 }
 
 __global__ void compact_rows(const int *row_ind,
                              int       *data_out,
                              int       *new_row_number,
                              int       *predicate,
                              int       *scatter_addres,
                              int        n) {
 
   int idx = threadIdx.x + blockDim.x * blockIdx.x;
 
   if(idx < n) {
     if(predicate[idx]) {
       int write_ind = scatter_addres[idx] - 1;
       int row = row_ind[idx];
       data_out[write_ind] = new_row_number[row]-1;
     }
   }
 }
 
 __global__ void vector_init_abs_kernel(const int *a,
                                        int       *b,
                                        int        n) {
 
   int i  = threadIdx.x + blockDim.x * blockIdx.x;
 
   if (i < n) {
     b[i] = abs(a[i]);
   }
 }
 
 
 /*******************************************************************************
  *                         Private Functions                                   *
  *******************************************************************************/
 
  /*
  *  Update the size of buffer used for the merge path based
  *  sparse matrix-vector product (spmv).
  */
 void update_mp_buffer(csr *P) {
 
   size_t bufferSizeInBytes = 0;
   float alpha = 1.0;
 
   checkCudaErrors(cusparseCsrmv_bufferSize(CUDA_handle->cusparseHandle,
                                            P->alg, P->m, P->n, P->nnz,
                                            &alpha,
                                            P->MatDescription, P->val, P->row_ptr, P->col_ind,
                                            NULL,
                                            &alpha,
                                            NULL,
                                            &bufferSizeInBytes));
   
   if (bufferSizeInBytes > P->bufferSizeInBytes) {
     cuda_free((void **) &P->buffer);                            
     cuda_malloc((void **) &P->buffer, bufferSizeInBytes);
     P->bufferSizeInBytes = bufferSizeInBytes;
   }
 }
 
  /*
  *  Creates a CSR matrix with the specified dimension (m,n,nnz).
  *  
  *  If specified, it allocates proper amount of device memory
  *  allocate_on_device = 1: device memory for CSR
  *  allocate_on_device = 2: device memory for CSR (+ col_ind)  
  */
 csr* csr_alloc(int m,
                int n,
                int nnz,
                int allocate_on_device) {
 
   csr *dev_mat = (csr*) calloc(1, sizeof(csr));
 
   if (!dev_mat) return NULL;
 
   dev_mat->m   = m;
   dev_mat->n   = n;
   dev_mat->nnz = nnz;
       
//  #ifdef IS_WINDOWS
//    /* MERGE_PATH is not working properly on WINDOWS */
//    dev_mat->alg = CUSPARSE_ALG_NAIVE;
//  #else
   dev_mat->alg = CUSPARSE_ALG_MERGE_PATH;
//  #endif
 
   dev_mat->buffer = NULL;
   dev_mat->bufferSizeInBytes = 0;
 
   checkCudaErrors(hipsparseCreateMatDescr(&dev_mat->MatDescription));
   hipsparseSetMatType(dev_mat->MatDescription, HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(dev_mat->MatDescription, HIPSPARSE_INDEX_BASE_ZERO);
 
   if (allocate_on_device > 0) {
     cuda_calloc((void **) &dev_mat->val, (dev_mat->nnz + 1) * sizeof(float));
     cuda_malloc((void **) &dev_mat->row_ptr, (dev_mat->m + 1) * sizeof(int)); 
     cuda_malloc((void **) &dev_mat->col_ind, dev_mat->nnz * sizeof(int));
 
     if (allocate_on_device > 1) {
       cuda_malloc((void **) &dev_mat->row_ind, dev_mat->nnz * sizeof(int));
     } 
   }
   return dev_mat;
 }
 
 /*
  *  Copy CSR matrix from host to device.
  *  The device memory should be pre-allocated.
  */
 void csr_copy_h2d(csr           *dev_mat,
                   const int   *h_row_ptr,
                   const int   *h_col_ind,
                   const float *h_val) {
 
   checkCudaErrors(hipMemcpy(dev_mat->row_ptr, h_row_ptr, (dev_mat->m + 1) * sizeof(int), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(dev_mat->col_ind, h_col_ind, dev_mat->nnz * sizeof(int), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(dev_mat->val, h_val, dev_mat->nnz * sizeof(float), hipMemcpyHostToDevice));
 }
 
 csr* csr_init(int          m,
               int          n,
               const int   *h_row_ptr,
               const int   *h_col_ind,
               const float *h_val) {
     
   csr *dev_mat = csr_alloc(m, n, h_row_ptr[m], 1);
   
   if (!dev_mat) return NULL;
   
   if (m == 0) return dev_mat;
 
   /* copy_matrix_to_device */
   csr_copy_h2d(dev_mat, h_row_ptr, h_col_ind, h_val);
   update_mp_buffer(dev_mat);
 
   return dev_mat;
 }
 
 /*
  *  Compress row indices from the COO format to the row pointer
  *  of the CSR format.
  */
 void compress_row_ind(csr *mat) {
 
   cuda_free((void** ) &mat->row_ptr);
   cuda_malloc((void** ) &mat->row_ptr, (mat->m + 1) * sizeof(float));
   checkCudaErrors(hipsparseXcoo2csr(CUDA_handle->cusparseHandle, mat->row_ind, mat->nnz, mat->m, mat->row_ptr, HIPSPARSE_INDEX_BASE_ZERO));
 }
 
 void csr_expand_row_ind(csr *mat) {
 
   if (!mat->row_ind) {
     cuda_malloc((void** ) &mat->row_ind, mat->nnz * sizeof(float));
     checkCudaErrors(hipsparseXcsr2coo(CUDA_handle->cusparseHandle, mat->row_ptr, mat->nnz, mat->m, mat->row_ind, HIPSPARSE_INDEX_BASE_ZERO));
   }
 }
 
 /*
  *  Sorts matrix in COO format by row. It returns a permutation
  *  vector that describes reordering of the elements.
  */
 int* coo_sort(csr *A) {
 
   int *A_to_At_permutation;
   char *pBuffer;
   size_t pBufferSizeInBytes;
 
   cuda_malloc((void **) &A_to_At_permutation, A->nnz * sizeof(int));
   checkCudaErrors(hipsparseCreateIdentityPermutation(CUDA_handle->cusparseHandle, A->nnz, A_to_At_permutation));
 
   checkCudaErrors(hipsparseXcoosort_bufferSizeExt(CUDA_handle->cusparseHandle, A->m, A->n, A->nnz, A->row_ind, A->col_ind, &pBufferSizeInBytes));
 
   cuda_malloc((void **) &pBuffer, pBufferSizeInBytes * sizeof(char));
 
   checkCudaErrors(hipsparseXcoosortByRow(CUDA_handle->cusparseHandle, A->m, A->n, A->nnz, A->row_ind, A->col_ind, A_to_At_permutation, pBuffer));
 
   cuda_free((void **) &pBuffer);
 
   return A_to_At_permutation;
 }
 
 /*
  * Compute transpose of a matrix in COO format.
  */
 void coo_tranpose(csr* A) {
   int m = A->m;
   A->m = A->n;
   A->n = m;
 
   int *row_ind = A->row_ind;
   A->row_ind = A->col_ind;
   A->col_ind = row_ind;
 }
 
 /*
  *  values[i] = values[permutation[i]] for i in [0,n-1]
  */
 void permute_vector(float     *values,
                     const int *permutation,
                     int        n) {
 
   float *permuted_values;
   cuda_malloc((void **) &permuted_values, n * sizeof(float));
 
   checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, n, values, permuted_values, permutation, HIPSPARSE_INDEX_BASE_ZERO));
 
   checkCudaErrors(hipMemcpy(values, permuted_values, n * sizeof(float), hipMemcpyDeviceToDevice));
   cuda_free((void **) &permuted_values);
 }
 
 /*
  *  target[i] = source[permutation[i]] for i in [0,n-1]
  *  
  *  target and source cannot point to the same location
  */
 void permute_vector(float       *target,
                     const float *source,
                     const int   *permutation,
                     int          n) {
 
   checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, n, source, target, permutation, HIPSPARSE_INDEX_BASE_ZERO));
 }
 
 /*
  *  Copy the values and pointers form target to the source matrix.
  *  The device memory of source has to be freed first to avoid a
  *  memory leak in case it holds allocated memory.
  *  
  *  The MatrixDescription has to be destroyed first since it is a
  *  pointer hidded by a typedef.
  *  
  *  The pointers of source matrix are set to NULL to avoid
  *  accidental freeing of the associated memory blocks.
  */
 void copy_csr(csr* target,
               csr* source) {
 
   target->m                 = source->m;
   target->n                 = source->n;
   target->nnz               = source->nnz;
   target->bufferSizeInBytes = source->bufferSizeInBytes;
   target->alg               = source->alg;
 
   hipsparseDestroyMatDescr(target->MatDescription);
   cuda_free((void **) &target->val);
   cuda_free((void **) &target->row_ind);
   cuda_free((void **) &target->row_ptr);
   cuda_free((void **) &target->col_ind);
   cuda_free((void **) &target->buffer);
 
   target->val            = source->val;
   target->row_ind        = source->row_ind;
   target->row_ptr        = source->row_ptr;
   target->col_ind        = source->col_ind;
   target->buffer         = source->buffer;
   target->MatDescription = source->MatDescription; 
 
   source->val            = NULL;
   source->row_ind        = NULL;
   source->row_ptr        = NULL;
   source->col_ind        = NULL;
   source->buffer         = NULL;
   source->MatDescription = NULL;
 }
 
 void csr_triu_to_full(csr    *P_triu,
                       int **P_triu_to_full_permutation,
                       int **P_diag_indices) {
 
   int number_of_blocks;
   int *has_non_zero_diag_element, *d_nnz_diag;
   int h_nnz_diag, Full_nnz, nnz_triu, n, nnz_max_Full;
   int offset;
 
   nnz_triu     = P_triu->nnz;
   n            = P_triu->n;
   nnz_max_Full = 2*nnz_triu + n;
 
   csr *Full_P = csr_alloc(n, n, nnz_max_Full, 2);
   cuda_calloc((void **) &has_non_zero_diag_element, n * sizeof(int));
   cuda_calloc((void **) &d_nnz_diag, sizeof(int));
 
   csr_expand_row_ind(P_triu);
 
   number_of_blocks = (nnz_triu / THREADS_PER_BLOCK) + 1;
   fill_full_matrix_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(Full_P->row_ind, Full_P->col_ind, d_nnz_diag, has_non_zero_diag_element, P_triu->row_ind, P_triu->col_ind, nnz_triu, n);
 
   offset = 2 * nnz_triu;
   number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
   add_diagonal_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(Full_P->row_ind + offset, Full_P->col_ind + offset, has_non_zero_diag_element, n);
 
   /* The Full matrix now is of size (2n)x(2n)
     *                  [P 0]
     *                  [0 D]
     * where P is the desired full matrix and D is
     * a diagonal that contains dummy values
   */
   
   checkCudaErrors(hipMemcpy(&h_nnz_diag, d_nnz_diag, sizeof(int), hipMemcpyDeviceToHost));
 
   Full_nnz = (2 * (nnz_triu - h_nnz_diag)) + n;
   int *d_P = coo_sort(Full_P);
 
   number_of_blocks = (nnz_triu / THREADS_PER_BLOCK) + 1;
   reduce_permutation_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(d_P, nnz_triu, Full_nnz);
 
   permute_vector(Full_P->val, P_triu->val, d_P, Full_nnz);
 
   cuda_malloc((void **) P_triu_to_full_permutation, Full_nnz * sizeof(int));
   checkCudaErrors(hipMemcpy(*P_triu_to_full_permutation, d_P, Full_nnz * sizeof(int), hipMemcpyDeviceToDevice));
   cuda_malloc((void **) P_diag_indices, n * sizeof(int));
 
   number_of_blocks = (Full_nnz / THREADS_PER_BLOCK) + 1;
   get_diagonal_indices_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(Full_P->row_ind, Full_P->col_ind, Full_nnz, *P_diag_indices);
 
   Full_P->nnz = Full_nnz;
   compress_row_ind(Full_P);
   update_mp_buffer(Full_P); 
   copy_csr(P_triu, Full_P);
 
   cuda_mat_free(Full_P);
   cuda_free((void **) &d_P);
   cuda_free((void **) &d_nnz_diag);
   cuda_free((void **) &has_non_zero_diag_element);
 }
 
 /**
  * Matrix A is converted from CSC to CSR. The data in A is interpreted as
  * being in CSC format, even if it is in CSR.
  * This operation is equivalent to a transpose. We temporarily allocate space
  * for the new matrix since this operation cannot be done inplace.
  * Additionally, a gather indices vector is generated to perform the conversion
  * from A to A' faster during a matrix update.
  */
 void csr_transpose(csr    *A,
                    int **A_to_At_permutation) {
 
   (*A_to_At_permutation) = NULL;
 
   if (A->nnz == 0) {
     int tmp = A->n;
     A->n = A->m;
     A->m = tmp;
     return;
   }
 
   csr_expand_row_ind(A);
   coo_tranpose(A);
   (*A_to_At_permutation) = coo_sort(A);
   compress_row_ind(A);
 
   permute_vector(A->val, *A_to_At_permutation, A->nnz);
 
   update_mp_buffer(A);
 }
 
 
 /*******************************************************************************
  *                           API Functions                                     *
  *******************************************************************************/
 
 void cuda_mat_init_P(const csc  *mat,
                      csr       **P,
                      float   **d_P_triu_val,
                      int     **d_P_triu_to_full_ind,
                      int     **d_P_diag_ind) {
 
   int n   = mat->n;
   int nnz = mat->p[n];
   
   /* Initialize upper triangular part of P */
   *P = csr_init(n, n, mat->p, mat->i, mat->x);
 
   /* Convert P to a full matrix. Store indices of diagonal and triu elements. */
   csr_triu_to_full(*P, d_P_triu_to_full_ind, d_P_diag_ind);
   csr_expand_row_ind(*P);
 
   /* We need 0.0 at val[nzz] -> nnz+1 elements */
   cuda_calloc((void **) d_P_triu_val, (nnz+1) * sizeof(float));
 
   /* Store triu elements */
   checkCudaErrors(hipMemcpy(*d_P_triu_val, mat->x, nnz * sizeof(float), hipMemcpyHostToDevice));
 }
 
 void cuda_mat_init_A(const csc  *mat,
                      csr       **A,
                      csr       **At,
                      int     **d_A_to_At_ind) {
 
   int m = mat->m;
   int n = mat->n;
 
   /* Initializing At is easy since it is equal to A in CSC */
   *At = csr_init(n, m, mat->p, mat->i, mat->x);
   csr_expand_row_ind(*At);
 
   /* We need to take transpose of At to get A */
   *A = csr_init(n, m, mat->p, mat->i, mat->x);
   csr_transpose(*A, d_A_to_At_ind);
   csr_expand_row_ind(*A);
 }
 
 void cuda_mat_update_P(const float  *Px,
                        const int    *Px_idx,
                        int           Px_n,
                        csr           **P,
                        float        *d_P_triu_val,
                        int          *d_P_triu_to_full_ind,
                        int          *d_P_diag_ind,
                        int           P_triu_nnz) {
 
   if (!Px_idx) { /* Update whole P */
     float *d_P_val_new;
 
     /* Allocate memory */
     cuda_malloc((void **) &d_P_val_new, (P_triu_nnz + 1) * sizeof(float));
 
     /* Copy new values from host to device */
     checkCudaErrors(hipMemcpy(d_P_val_new, Px, P_triu_nnz * sizeof(float), hipMemcpyHostToDevice));
 
     checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, (*P)->nnz, d_P_val_new, (*P)->val, d_P_triu_to_full_ind, HIPSPARSE_INDEX_BASE_ZERO));
 
     cuda_free((void **) &d_P_val_new);
   }
   else { /* Update P partially */
     float *d_P_val_new;
     int   *d_P_ind_new;
 
     /* Allocate memory */
     cuda_malloc((void **) &d_P_val_new, Px_n * sizeof(float));
     cuda_malloc((void **) &d_P_ind_new, Px_n * sizeof(int));
 
     /* Copy new values and indices from host to device */
     checkCudaErrors(hipMemcpy(d_P_val_new, Px,     Px_n * sizeof(float), hipMemcpyHostToDevice));
     checkCudaErrors(hipMemcpy(d_P_ind_new, Px_idx, Px_n * sizeof(int),   hipMemcpyHostToDevice));
 
     /* Update d_P_triu_val */
     scatter(d_P_triu_val, d_P_val_new, d_P_ind_new, Px_n);
 
     /* Gather from d_P_triu_val to update full P */
     checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, (*P)->nnz, d_P_triu_val, (*P)->val, d_P_triu_to_full_ind, HIPSPARSE_INDEX_BASE_ZERO));
 
     cuda_free((void **) &d_P_val_new);
     cuda_free((void **) &d_P_ind_new);
   }
 }
 
 void cuda_mat_update_A(const float  *Ax,
                        const int    *Ax_idx,
                        int           Ax_n,
                        csr           **A,
                        csr           **At,
                        int          *d_A_to_At_ind) {
 
   int Annz     = (*A)->nnz;
   float *Aval  = (*A)->val;
   float *Atval = (*At)->val;
 
   if (!Ax_idx) { /* Update whole A */
     /* Updating At is easy since it is equal to A in CSC */
     checkCudaErrors(hipMemcpy(Atval, Ax, Annz * sizeof(float), hipMemcpyHostToDevice));
 
     /* Updating A requires transpose of A_new */
     checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, Annz, Atval, Aval, d_A_to_At_ind, HIPSPARSE_INDEX_BASE_ZERO));
   }
   else { /* Update A partially */
     float *d_At_val_new;
     int   *d_At_ind_new;
 
     /* Allocate memory */
     cuda_malloc((void **) &d_At_val_new, Ax_n * sizeof(float));
     cuda_malloc((void **) &d_At_ind_new, Ax_n * sizeof(int));
 
     /* Copy new values and indices from host to device */
     checkCudaErrors(hipMemcpy(d_At_val_new, Ax,     Ax_n * sizeof(float), hipMemcpyHostToDevice));
     checkCudaErrors(hipMemcpy(d_At_ind_new, Ax_idx, Ax_n * sizeof(int),   hipMemcpyHostToDevice));
 
     /* Update At first since it is equal to A in CSC */
     scatter(Atval, d_At_val_new, d_At_ind_new, Ax_n);
 
     cuda_free((void **) &d_At_val_new);
     cuda_free((void **) &d_At_ind_new);
 
     /* Gather from Atval to construct Aval */
     checkCudaErrors(cusparseTgthr(CUDA_handle->cusparseHandle, Annz, Atval, Aval, d_A_to_At_ind, HIPSPARSE_INDEX_BASE_ZERO));
   }
 }
 
 void cuda_mat_free(csr *mat) {
   if (mat) {
     cuda_free((void **) &mat->val);
     cuda_free((void **) &mat->row_ptr);
     cuda_free((void **) &mat->col_ind);
     cuda_free((void **) &mat->buffer);
     cuda_free((void **) &mat->row_ind);
     hipsparseDestroyMatDescr(mat->MatDescription);
     free(mat);
   }
 }
 
 void cuda_submat_byrows(const csr    *A,
                         const int  *d_rows,
                         csr         **Ared,
                         csr         **Aredt) {
 
   int new_m = 0;
 
   int n   = A->n;
   int m   = A->m;
   int nnz = A->nnz;
 
   int *d_predicate;
   int *d_compact_address;
   int *d_row_predicate;
   int *d_new_row_number;
 
   cuda_malloc((void **) &d_row_predicate,  m * sizeof(int));
   cuda_malloc((void **) &d_new_row_number, m * sizeof(int));
 
   cuda_malloc((void **) &d_predicate,       nnz * sizeof(int));
   cuda_malloc((void **) &d_compact_address, nnz * sizeof(int));
 
   // Copy rows array to device and set -1s to ones
   checkCudaErrors(hipMemcpy(d_row_predicate, d_rows, m * sizeof(int), hipMemcpyDeviceToDevice));
   vector_init_abs_kernel<<<(m/THREADS_PER_BLOCK) + 1,THREADS_PER_BLOCK>>>(d_row_predicate, d_row_predicate, m);
 
   // Calculate new row numbering and get new number of rows
   thrust::inclusive_scan(thrust::device, d_row_predicate, d_row_predicate + m, d_new_row_number);
   if (m) {
     checkCudaErrors(hipMemcpy(&new_m, &d_new_row_number[m-1], sizeof(int), hipMemcpyDeviceToHost));
   }
   else {
     (*Ared) = (csr *) calloc(1, sizeof(csr));
     (*Ared)->n = n;
 
     (*Aredt) = (csr *) calloc(1, sizeof(csr));
     (*Aredt)->m = n;
 
     return;
   }
 
   // Generate predicates per element from per row predicate
   predicate_generator_kernel<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->row_ind, d_row_predicate, d_predicate, nnz);
 
   // Get array offset for compacting and new nnz
   thrust::inclusive_scan(thrust::device, d_predicate, d_predicate + nnz, d_compact_address);
   int nnz_new;
   if (nnz) checkCudaErrors(hipMemcpy(&nnz_new, &d_compact_address[nnz-1], sizeof(int), hipMemcpyDeviceToHost));
 
   // allocate new matrix (2 -> allocate row indices as well)
   (*Ared) = csr_alloc(new_m, n, nnz_new, 2);
 
   // Compact arrays according to given predicates, special care has to be taken for the rows
   compact_rows<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->row_ind, (*Ared)->row_ind, d_new_row_number, d_predicate, d_compact_address, nnz);
   compact<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->col_ind, (*Ared)->col_ind, d_predicate, d_compact_address, nnz);
   compact<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->val, (*Ared)->val, d_predicate, d_compact_address, nnz);
 
   // Generate row pointer
   compress_row_ind(*Ared);
 
   // Update merge path buffer (CsrmvEx)
   update_mp_buffer(*Ared);
 
   // We first make a copy of Ared
   *Aredt = csr_alloc(new_m, n, nnz_new, 1);
   checkCudaErrors(hipMemcpy((*Aredt)->val,     (*Ared)->val,     nnz_new   * sizeof(float), hipMemcpyDeviceToDevice));
   checkCudaErrors(hipMemcpy((*Aredt)->row_ptr, (*Ared)->row_ptr, (new_m+1) * sizeof(int),   hipMemcpyDeviceToDevice));
   checkCudaErrors(hipMemcpy((*Aredt)->col_ind, (*Ared)->col_ind, nnz_new   * sizeof(int),   hipMemcpyDeviceToDevice));
 
   int *d_A_to_At_ind;
   csr_transpose(*Aredt, &d_A_to_At_ind);
 
   // Update merge path buffer (CsrmvEx)
   update_mp_buffer(*Aredt);
 
   cuda_free((void**)&d_A_to_At_ind);
   cuda_free((void**)&d_predicate);
   cuda_free((void**)&d_compact_address);
   cuda_free((void**)&d_row_predicate);
   cuda_free((void**)&d_new_row_number);
 }
 
 void cuda_mat_get_m(const csr *mat,
                     int     *m) {
 
   (*m) = mat->m;
 }
 
 void cuda_mat_get_n(const csr *mat,
                     int     *n) {
 
   (*n) = mat->n;
 }
 
 void cuda_mat_get_nnz(const csr *mat,
                       int     *nnz) {
 
   (*nnz) = mat->nnz;
 }